#include "hip/hip_runtime.h"
#include "helpers.h"
#include "matrix2d.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <cassert>
#include <cstring>
#include <iostream>

namespace cg = cooperative_groups;

template <class T>
struct SharedMemory
{
	__device__ inline operator T*()
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}

	__device__ inline operator const T*() const
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
};

__global__ void cuda_add(float* mat1, float* mat2, float* mat3, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat3[i] = mat1[i] + mat2[i];
}

__global__ void cuda_add_self(float* mat1, float* mat2, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat1[i] += mat2[i];
}

__global__ void cuda_sub(float* mat1, float* mat2, float* mat3, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat3[i] = mat1[i] - mat2[i];
}

__global__ void cuda_sub_self(float* mat1, float* mat2, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat1[i] -= mat2[i];
}

__global__ void cuda_mul(float* mat1, float* mat2, float* mat3, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat3[i] = mat1[i] * mat2[i];
}

__global__ void cuda_mul_self(float* mat1, float* mat2, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat1[i] *= mat2[i];
}

__global__ void cuda_mat_mul(float* mat1, float* mat2, float* mat3, int nx, int ny, int m)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int nxy = nx * y;
	float dot_prod = 0.0F;
	for (int i = 0; i < m; ++i) { dot_prod += mat1[nxy + i] * mat2[nx * i + x]; }

	mat3[nx * y + x] = dot_prod;
}

__global__ void cuda_add_scalar(float* mat1, float* mat2, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat2[i] = mat1[i] + scalar;
}

__global__ void cuda_add_scalar_self(float* mat, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat[i] += scalar;
}

__global__ void cuda_sub_scalar(float* mat1, float* mat2, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat2[i] = mat1[i] - scalar;
}

__global__ void cuda_sub_scalar_self(float* mat, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat[i] -= scalar;
}

__global__ void cuda_mul_scalar(float* mat1, float* mat2, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat2[i] = mat1[i] * scalar;
}

__global__ void cuda_mul_scalar_self(float* mat, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	const int i = nx * y + x;
	mat[i] *= scalar;
}

__global__ void cuda_sum(float* mat, float* sum, int n)
{
	// Handle to thread block group
	cg::thread_block cta = cg::this_thread_block();
	float* s_data = SharedMemory<float>();

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	float tsum = (i < n) ? mat[i] : 0.0F;
	s_data[tid] = tsum;
	cg::sync(cta);

	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s) { s_data[tid] = tsum = tsum + s_data[tid + s]; }

		cg::sync(cta);
	}

	if (tid == 0) { atomicAdd(sum, tsum); }
}

__global__ void cuda_fill(float* mat, float scalar, int nx, int ny)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x >= nx) || (y >= ny)) { return; }
	mat[nx * y + x] = scalar;
}

Matrix2d::Matrix2d(Size size) : size_(size), num_elements_(size_.x * size_.y)
{
	assert(size_.x > 0);
	assert(size_.y > 0);
	blocks_ = dim3(
		std::min<unsigned int>(64, (size_.x + threads_.x - 1) / threads_.x),
		std::min<unsigned int>(64, (size_.y + threads_.y - 1) / threads_.y));

	CHECK_CUDA_ERROR(hipMalloc(&d_data_, num_elements_ * sizeof(float)));
}

Matrix2d::Matrix2d(const std::vector<float>& data, Size size)
{
	assert(data.size() > 0);
	if (size.y < 0)
	{
		size.x = 1;
		size.y = static_cast<int>(data.size());
	}
	else
	{
		assert(size.x > 0);
		assert(size.y > 0);
		assert(data.size() == static_cast<size_t>(size.x * size.y));
	}

	blocks_ = dim3(
		std::min<unsigned int>(64, (size.x + threads_.x - 1) / threads_.x),
		std::min<unsigned int>(64, (size.y + threads_.y - 1) / threads_.y));

	CHECK_CUDA_ERROR(hipMalloc((void**)&d_data_, data.size() * sizeof(float)));
	set(data, size);
}

Matrix2d::Matrix2d(Size size, float scalar) : size_(size), num_elements_(size_.x * size_.y)
{
	assert(size_.x > 0);
	assert(size_.y > 0);
	blocks_ = dim3(
		std::min<unsigned int>(64, (size_.x + threads_.x - 1) / threads_.x),
		std::min<unsigned int>(64, (size_.y + threads_.y - 1) / threads_.y));

	CHECK_CUDA_ERROR(hipMalloc(&d_data_, num_elements_ * sizeof(float)));
	fill(scalar);
}

Matrix2d::Matrix2d(const Matrix2d& mat)
		: size_(mat.size_), num_elements_(mat.num_elements_), blocks_(mat.blocks_), threads_(mat.threads_)
{
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_data_, num_elements_ * sizeof(float)));
	CHECK_CUDA_ERROR(hipMemcpy(d_data_, mat.d_data_, num_elements_ * sizeof(float), hipMemcpyDeviceToDevice));
}

Matrix2d::Matrix2d(Matrix2d&& mat)
		: d_data_(mat.d_data_)
		, size_(mat.size_)
		, num_elements_(mat.num_elements_)
		, blocks_(mat.blocks_)
		, threads_(mat.threads_)
{
	mat.d_data_ = nullptr;
	mat.size_ = {0, 0};
	mat.num_elements_ = 0;
}

Matrix2d::~Matrix2d()
{
	CHECK_CUDA_ERROR(hipFree(d_data_));
	num_elements_ = 0;
}

Matrix2d& Matrix2d::operator=(const Matrix2d& mat)
{
	check_bounds_match(mat);
	assert(d_data_ != nullptr);
	CHECK_CUDA_ERROR(hipMemcpy(d_data_, mat.d_data_, num_elements_ * sizeof(float), hipMemcpyDeviceToDevice));
	return *this;
}

Matrix2d& Matrix2d::operator=(Matrix2d&& mat)
{
	size_ = mat.size_;
	num_elements_ = mat.num_elements_;
	blocks_ = mat.blocks_;
	threads_ = mat.threads_;
	CHECK_CUDA_ERROR(hipFree(d_data_));
	d_data_ = mat.d_data_;
	mat.d_data_ = nullptr;
	return *this;
}

Size Matrix2d::size() const
{
	return size_;
}

int Matrix2d::num_elements() const
{
	return num_elements_;
}

void Matrix2d::check_bounds_match(const Matrix2d& mat) const
{
	if (mat.size_.x != size_.x)
	{
		std::cerr << "x dims do not match. Expected " << size_.x << " but got " << mat.size_.x << std::endl;
		throw std::runtime_error("Dim mismatch");
	}
	if (mat.size_.y != size_.y)
	{
		// y dims to not match
		std::cerr << "y dims do not match. Expected " << size_.y << " but got " << mat.size_.y << std::endl;
		throw std::runtime_error("Dim mismatch");
	}
}

Matrix2d Matrix2d::add(const Matrix2d& mat) const
{
	check_bounds_match(mat);

	Matrix2d mat_result(size_);

	KERNEL_CALL(cuda_add, d_data_, mat.d_data_, mat_result.d_data_, size_.x, size_.y);

	return mat_result;
}

Matrix2d Matrix2d::sub(const Matrix2d& mat) const
{
	check_bounds_match(mat);

	Matrix2d mat_result(size_);

	KERNEL_CALL(cuda_sub, d_data_, mat.d_data_, mat_result.d_data_, size_.x, size_.y);

	return mat_result;
}

Matrix2d Matrix2d::mul(const Matrix2d& mat) const
{
	if (size_.x == mat.size_.y)
	{
		Matrix2d mat_result({size_.y, mat.size_.x});

		KERNEL_CALL(cuda_mat_mul, d_data_, mat.d_data_, mat_result.d_data_, mat.size_.x, size_.y, size_.x);

		return mat_result;
	}
	else
	{
		std::cerr << "x dim of mat1 does not match y dim of mat2. Expected " << size_.x << " but got " << mat.size_.y
							<< std::endl;
		throw std::runtime_error("Dim mismatch");
	}
}

Matrix2d Matrix2d::add(const float scalar) const
{
	Matrix2d mat_result(size_);

	KERNEL_CALL(cuda_add_scalar, d_data_, mat_result.d_data_, scalar, size_.x, size_.y);

	return mat_result;
}

Matrix2d Matrix2d::sub(const float scalar) const
{
	Matrix2d mat_result(size_);

	KERNEL_CALL(cuda_sub_scalar, d_data_, mat_result.d_data_, scalar, size_.x, size_.y);

	return mat_result;
}

Matrix2d Matrix2d::mul(const float scalar) const
{
	Matrix2d mat_result(size_);

	KERNEL_CALL(cuda_mul_scalar, d_data_, mat_result.d_data_, scalar, size_.x, size_.y);

	return mat_result;
}

Matrix2d Matrix2d::operator+(const Matrix2d& mat) const
{
	return add(mat);
}

Matrix2d& Matrix2d::operator+=(const Matrix2d& mat)
{
	check_bounds_match(mat);

	KERNEL_CALL(cuda_add_self, d_data_, mat.d_data_, size_.x, size_.y);

	return *this;
}

Matrix2d Matrix2d::operator-(const Matrix2d& mat) const
{
	return sub(mat);
}

Matrix2d& Matrix2d::operator-=(const Matrix2d& mat)
{
	check_bounds_match(mat);

	KERNEL_CALL(cuda_sub_self, d_data_, mat.d_data_, size_.x, size_.y);

	return *this;
}

Matrix2d Matrix2d::operator*(const Matrix2d& mat) const
{
	return mul(mat);
}

Matrix2d& Matrix2d::operator*=(const Matrix2d& mat)
{
	check_bounds_match(mat);

	KERNEL_CALL(cuda_mul_self, d_data_, mat.d_data_, size_.x, size_.y);

	return *this;
}

Matrix2d Matrix2d::operator+(const float scalar) const
{
	return add(scalar);
}

Matrix2d& Matrix2d::operator+=(const float scalar)
{
	KERNEL_CALL(cuda_add_scalar_self, d_data_, scalar, size_.x, size_.y);

	return *this;
}

Matrix2d Matrix2d::operator-(const float scalar) const
{
	return sub(scalar);
}

Matrix2d& Matrix2d::operator-=(const float scalar)
{
	KERNEL_CALL(cuda_sub_scalar_self, d_data_, scalar, size_.x, size_.y);

	return *this;
}

Matrix2d Matrix2d::operator*(const float scalar) const
{
	return mul(scalar);
}

Matrix2d& Matrix2d::operator*=(const float scalar)
{
	KERNEL_CALL(cuda_mul_scalar_self, d_data_, scalar, size_.x, size_.y);

	return *this;
}

float Matrix2d::sum() const
{
	float* d_sum;
	CHECK_CUDA_ERROR(hipMallocManaged((void**)&d_sum, sizeof(float)));
	const dim3 dim_block(threads_.x * threads_.y * threads_.z, 1, 1);
	const dim3 dim_grid(blocks_.x * blocks_.y * blocks_.z, 1, 1);
	const int smem_size = (dim_block.x <= 32) ? 2 * dim_block.x * sizeof(float) : dim_block.x * sizeof(float);
	cuda_sum<<<dim_grid, dim_block, smem_size>>>(d_data_, d_sum, num_elements_);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR(hipGetLastError());
	float sum = *d_sum;
	CHECK_CUDA_ERROR(hipFree(d_sum));
	return sum;
}

void Matrix2d::fill(float scalar)
{
	KERNEL_CALL(cuda_fill, d_data_, scalar, size_.x, size_.y);
}

float Matrix2d::get(int y, int x) const
{
	float element;
	int i = size_.x * y + x;
	CHECK_CUDA_ERROR(hipMemcpy(&element, &(d_data_[i]), sizeof(float), hipMemcpyDeviceToHost));
	return element;
}

std::vector<float> Matrix2d::get() const
{
	std::vector<float> data;
	data.resize(num_elements_);
	CHECK_CUDA_ERROR(hipMemcpy(data.data(), d_data_, data.size() * sizeof(float), hipMemcpyDeviceToHost));
	return data;
}

void Matrix2d::set(const std::vector<float>& data, Size size)
{
	assert(size.x > 0);
	assert(size.y > 0);
	size_ = size;
	num_elements_ = static_cast<int>(data.size());
	CHECK_CUDA_ERROR(hipMemcpy(d_data_, data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR(hipGetLastError());
}
